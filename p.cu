#include "hip/hip_runtime.h"
#include <torch/types.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cfloat>

template<const int num_threads_x,
        const int num_threads_y,
        const int N,
        const int d,
        const int b_r,
        const int b_c
>
__global__ void fa(float* Q, float* K, float* V, float* O, float* L, float* M){
    /*
    parameter: name base on paper

    m in N
    n in d_k
    k in d_v

    o_size = N x d

    q_size = N x d
    k_size = N x d
    v_size = N x d
    */

    //SRAM allocation step 3 in paper
    __shared__ float q[b_r][d];
    __shared__ float k[b_c][d];
    __shared__ float v[b_c][d];
    int T_r =  (int)(N / b_r); // divide operations are slow in GPU find a way to replace it
    int T_c =  (int)(N / b_c);
    //SRAM allocation step 4 in paper
    __shared__ float o[b_r][d];
    __shared__ float s[b_r][b_c];
    __shared__ float l[b_r];
    __shared__ float m[b_r];
    float scale = 1.0f / sqrtf((float)d);
    //calculate the thread num in block and the thread num in row and col
    int thread_num_inblock = num_threads_x*num_threads_y;
    unsigned int b_idx = threadIdx.x + threadIdx.y*blockDim.x;// 2D matrix but 1D idx
    int data_num_block_q = d * b_r;//for q and o
    int data_num_block_k = d * b_c;//for k v
    //assumption that is one data for one thread and b_r == b_c
   
    //load gmem to smem
    for(int j = 0;j < T_c;j++){//step 5
        for(int idx = b_idx;idx < data_num_block_k;idx += blockDim.x * blockDim.y){
            
            v[idx / d][idx % d] = V[idx  + j * data_num_block_k];//each block load its own gmem to smem
            k[idx / d][idx % d] = K[idx  + j * data_num_block_k];//in paper it is step 6
        }
        __syncthreads();
        for(int iter = 0;iter < T_r;iter++){//step 7
           for (int i = b_idx; i < b_r * d; i += blockDim.x * blockDim.y) {
                int row = i / d;
                int col = i % d;
                q[row][col] = Q[iter * b_r * d + i];
                o[row][col] = O[iter * b_r * d + i];
            }

            for(int idx = b_idx;idx < b_r;idx += blockDim.x * blockDim.y){
                l[idx] = L[idx + iter*b_r];
                m[idx] = M[idx + iter*b_r];
            }
            __syncthreads();
            //mat multiply
            //semm step 9
            int tx = threadIdx.x;
            int ty = threadIdx.y;
           
            for (int row = ty; row < b_r; row += blockDim.y) {
                for (int col = tx; col < b_c; col += blockDim.x) {
                    float sum = 0.0f;
                    for (int k_dim = 0; k_dim < d; k_dim++) {
                        sum += q[row][k_dim] * k[col][k_dim];
                    }
                    // *** BUG FIX 1: 增加了缩放步骤 ***
                    s[row][col] = sum * scale;
                }
            }
            __syncthreads(); // Essential: All threads must finish computing their part of 's'
           
            //step 10
            //calculate the max
            __shared__ float m_up[b_r];
            __shared__ float l_up[b_r];
            if (tx == 0) {
                for (int row = ty; row < b_r; row += blockDim.y) {
                    // 1. 找最大值 m_up
                    float row_max = -FLT_MAX;
                    for (int col = 0; col < b_c; col++) {
                        if (s[row][col] > row_max) {
                            row_max = s[row][col];
                        }
                    }
                    m_up[row] = row_max;

                    // 2. 计算 P_ij 并求和得到 l_up
                    float row_sum_exp = 0.0f;
                    for (int col = 0; col < b_c; col++) {
                        float p_val = __expf(s[row][col] - row_max);
                        s[row][col] = p_val; // 将 s 矩阵原地更新为 P 矩阵
                        row_sum_exp += p_val;
                    }
                    l_up[row] = row_sum_exp;
                }
            }
            __syncthreads();

            //step 11
            __shared__ float m_new[b_r];
            __shared__ float l_new[b_r];
            if (tx == 0) {
                for (int row = ty; row < b_r; row += blockDim.y) {
                    m_new[row] = fmaxf(m[row], m_up[row]);
                    l_new[row] = __expf(m[row] - m_new[row]) * l[row] + __expf(m_up[row] - m_new[row]) * l_up[row];
                }
            }
            __syncthreads();
            //calculate o_i
            __shared__ float pv[b_r][d];
            // Each thread (ty, tx) will be responsible for computing one element pv[ty][tx].
            // This requires iterating through the inner dimension 'b_c'.

            for (int row = ty; row < b_r; row += blockDim.y) {
                for (int col = tx; col < d; col += blockDim.x) {
                    float pv_sum = 0.0f;
                    for (int k_dim = 0; k_dim < b_c; k_dim++) {
                        pv_sum += s[row][k_dim] * v[k_dim][col];
                    }
                    pv[row][col] = pv_sum;
                }
            }
            __syncthreads();
            for (int row = ty; row < b_r; row += blockDim.y) {
                float m_old = m[row];
                float l_old = l[row];
                float m_new_val = m_new[row];
                float l_new_val = l_new[row];

                for (int col = tx; col < d; col += blockDim.x) {
                    float o_old = o[row][col];
                    float pv_val = pv[row][col];
                    // 更新公式
                    o[row][col] = (l_old * __expf(m_old - m_new_val) * o_old + __expf(m_up[row] - m_new_val) * pv_val) / l_new_val;
                }
            }
             __syncthreads();
             if (tx == 0) {
                 for (int row = ty; row < b_r; row += blockDim.y) {
                    l[row] = l_new[row];
                    m[row] = m_new[row];
                 }
            }
            __syncthreads();

            for (int i = b_idx; i < b_r * d; i += blockDim.x * blockDim.y) {
                O[iter * b_r * d + i] = o[i / d][i % d];
            }
            for (int i = b_idx; i < b_r; i += blockDim.x * blockDim.y) {
                L[iter * b_r + i] = l[i];
                M[iter * b_r + i] = m[i];
            }
            __syncthreads();
        }
        
    }
}

torch::Tensor forward(torch::Tensor Q, torch::Tensor K, torch::Tensor V) {
    // 确定内核参数
    const int num_threads_x = 512;
    const int num_threads_y = 2;
    const int b_r = 4;
    const int b_c = 4;
    
    // 获取输入尺寸
    const int B = Q.size(0); const int nh = Q.size(1);
    const int N = Q.size(2); const int d = Q.size(3);
    
    // 初始化输出和中间张量
    auto O = torch::zeros_like(Q);
    auto L = torch::zeros({B, nh, N});
    auto M = torch::full({B, nh, N}, -INFINITY);
    torch::Device device(torch::kCUDA);
    L = L.to(device); M = M.to(device);
    
    // 打印SRAM使用信息
    const int sram_size = (b_r * d * sizeof(float)) + (b_c * d * sizeof(float)) + 
                         (b_c * d * sizeof(float)) + (b_r * d * sizeof(float)) + 
                         (b_r * b_c * sizeof(float)) + (b_r * sizeof(float)) + 
                         (b_r * sizeof(float));
    int max_sram_size;
    hipDeviceGetAttribute(&max_sram_size, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);
    printf("最大共享内存: %d字节, 请求共享内存: %d字节\n", max_sram_size, sram_size);
    
    // 设置CUDA内核启动参数
    dim3 grid_dim(B, nh);  // batch_size x num_heads
    dim3 block_dim(num_threads_x, num_threads_y);
    
    // 为每个批次和头部启动内核
    for (int b = 0; b < B; b++) {
        for (int h = 0; h < nh; h++) {
            // 获取当前批次和头部的数据指针
            float* q_ptr = Q.index({b, h}).data_ptr<float>();
            float* k_ptr = K.index({b, h}).data_ptr<float>();
            float* v_ptr = V.index({b, h}).data_ptr<float>();
            float* o_ptr = O.index({b, h}).data_ptr<float>();
            float* l_ptr = L.index({b, h}).data_ptr<float>();
            float* m_ptr = M.index({b, h}).data_ptr<float>();
            
            // 启动fa内核
            fa<num_threads_x, num_threads_y, 2048, 512, b_r, b_c><<<1, block_dim>>>(
                q_ptr, k_ptr, v_ptr, o_ptr, l_ptr, m_ptr
            );
        }
    }
    
    // 检查CUDA错误
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA错误: %s\n", hipGetErrorString(error));
    }
    
    return O;
} 